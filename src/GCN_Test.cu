#include "hip/hip_runtime.h"
#include <stdio.h>
//#include <stdint.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <gmp.h>
#include <cgbn/cgbn.h>
#include <utility/cpu_support.h>
#include <utility/cpu_simple_bn_math.h>
#include <utility/gpu_support.h>

#include "../TimeMeasuring.hpp"

// IMPORTANT:  DO NOT DEFINE TPI OR BITS BEFORE INCLUDING CGBN
#define TPI 32
#define BITS 256
#define INSTANCES 10000000

struct instance_t {
  cgbn_mem_t<BITS> a;
  cgbn_mem_t<BITS> b;
  cgbn_mem_t<BITS> m;
};

// helpful typedefs for the kernel
typedef cgbn_context_t<TPI> context_t;
typedef cgbn_env_t<context_t, BITS> env_t;

// the actual kernel
__global__ void kernel_add(cgbn_error_report_t *report, instance_t* instances, cgbn_mem_t<BITS>* results, uint32_t count) {
  // decode an instance number from the blockIdx and threadIdx
  int32_t instance=(blockIdx.x*blockDim.x + threadIdx.x)/TPI;
  if(instance>=count)
    return;

  context_t      bn_context(cgbn_report_monitor, report, instance);   // construct a context
  env_t          bn_env(bn_context.env<env_t>());                     // construct an environment for 1024-bit math
  env_t::cgbn_t  a, b, r, m;                                          // define a, b, r as 1024-bit bignums

  cgbn_load(bn_env, a, &(instances[instance].a));      // load my instance's a value
  cgbn_load(bn_env, b, &(instances[instance].b));      // load my instance's b value
  cgbn_load(bn_env, m, &(instances[instance].m));      // load my instance's b value

  // convert a and b to Montgomery space
  uint32_t np0 = cgbn_bn2mont(bn_env, a, a, m);
  cgbn_bn2mont(bn_env, b, b, m);

  cgbn_mont_mul(bn_env, r, a, b, m, np0); // a * b mod n

  // convert r back to normal space
  cgbn_mont2bn(bn_env, r, r, m, np0);

  cgbn_store(bn_env, &(results[instance]), r); // store r into mul
}

extern "C" void a_x_b_mod_m(uint8_t const* data, uint8_t* result, uint32_t count) {
  TAKEN_TIME();
  CUDA_CHECK(hipSetDevice(0));

  cgbn_error_report_t *report;
  CUDA_CHECK(cgbn_error_report_alloc(&report));

  instance_t *gpuInstances;
  cgbn_mem_t<BITS> *gpuResults;
  CUDA_CHECK(hipMalloc((void **)&gpuInstances, sizeof(instance_t)*count));
  CUDA_CHECK(hipMalloc((void **)&gpuResults, sizeof(cgbn_mem_t<BITS>)*count));
  CUDA_CHECK(hipMemcpy(gpuInstances, data, sizeof(instance_t)*count, hipMemcpyHostToDevice));

  // launch with 32 threads per instance, 128 threads (4 instances) per block
  kernel_add<<<(INSTANCES + 3) / 4, 128>>>(report, gpuInstances, gpuResults, count);

  // error report uses managed memory, so we sync the device (or stream) and check for cgbn errors
  CUDA_CHECK(hipDeviceSynchronize());
  CGBN_CHECK(report);

  // copy the instances back from gpuMemory
  CUDA_CHECK(hipMemcpy(result, gpuResults, sizeof(cgbn_mem_t<BITS>) * count, hipMemcpyDeviceToHost));

  CUDA_CHECK(hipFree(gpuInstances));
  CUDA_CHECK(hipFree(gpuResults));
  CUDA_CHECK(cgbn_error_report_free(report));
}

